#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "common.h"

constexpr size_t BLOCK_SIZE = 512;

unsigned int total_threads;

thrust::host_vector<double> host_m;
thrust::host_vector<Point> host_pos;

void generate_data_cuda(thrust::host_vector<double>& host_m, thrust::host_vector<Point>& host_pos,
                        const int n) {
    // initialize a random distribution
    std::random_device dev;
    std::mt19937 rng(dev());
    std::uniform_int_distribution<std::mt19937::result_type> uniform(0, RAND_MAX);
    std::uniform_int_distribution<std::mt19937::result_type> u_mass(1, MAX_MASS);

    for (int i = 0; i < n; i++) {
        host_m[i] = u_mass(rng);
        host_pos[i].x = 2000.0 + uniform(rng) % (BOUND_X / 4);
        host_pos[i].y = 2000.0 + uniform(rng) % (BOUND_Y / 4);
    }
}

__device__ Force get_force_cuda(const double m1, const double m2, const Point& pos1,
                                const Point& pos2) {
    const double dx = pos2.x - pos1.x, dy = pos2.y - pos1.y;
    const double r_sqr = dx * dx + dy * dy;
    const double r = sqrt(r_sqr);
    const double f = GRAVITY_CONST * m1 * m2 / (r_sqr + FLOAT_OP_ERROR);
    return Force{f * dx / r, f * dy / r};
}

__device__ void handle_wall_collision_cuda(Point& pos, Velocity& v) {
    if (pos.x < 0) {
        pos.x = 0;
        v.x = -v.x;
    }
    else if (pos.x > BOUND_X) {
        pos.x = BOUND_X;
        v.x = -v.x;
    }
    if (pos.y < 0) {
        pos.y = 0;
        v.y = -v.y;
    }
    else if (pos.y > BOUND_Y) {
        pos.y = BOUND_Y;
        v.y = -v.y;
    }
}

__global__ void update_positions(const double* m, Point* pos, Velocity* v, const int n) {
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        // X = X + V * dt
        pos[idx] += v[idx] * DT;
        // handle wall collision
        handle_wall_collision_cuda(pos[idx], v[idx]);
        // handle collision with other bodies and bounce
        for (int j = 0; j < n; ++j) {
            if (idx == j) continue;
            if (pos[idx].sqr_dist(pos[j]) < COLLISION_DIST2) {
                // do bounce
                Velocity& v1 = v[idx];
                Point& pos1 = pos[idx];
                // simple workaround: invert the speed
                v1.x = -v1.x;
                v1.y = -v1.y;
                // move the body away
                pos1 += v1 * DT;
            }
        }
    }
}

__global__ void update_velocities(const double* m, const Point* pos, Velocity* v, const int n) {
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        Force f{};
        // sum up all the forces
        for (int j = 0; j < n; ++j) {
            if (idx == j) continue;
            f += get_force_cuda(m[idx], m[j], pos[idx], pos[j]);
        }
        // v += at
        v[idx] += (f / m[idx]) * DT;
    }
}

void master() {
    using namespace std::chrono;

    host_m.resize(n_body);
    host_pos.resize(n_body);

    generate_data_cuda(host_m, host_pos, n_body);

    // copy data from host to device
    thrust::device_vector<double> device_m = host_m;
    thrust::device_vector<Point> device_pos = host_pos;
    thrust::device_vector<Velocity> device_v(n_body);

    const unsigned int n_block =
        n_body % BLOCK_SIZE == 0 ? n_body / BLOCK_SIZE : n_body / BLOCK_SIZE + 1;

    // split the data to arrange jobs
    total_threads = n_block * BLOCK_SIZE;

    for (int i = 0; i < n_iteration; i++) {
        high_resolution_clock::time_point t1 = high_resolution_clock::now();

        update_velocities<<<n_block, BLOCK_SIZE>>>(raw_pointer_cast(device_m.data()),
                                                   raw_pointer_cast(device_pos.data()),
                                                   raw_pointer_cast(device_v.data()), n_body);

        update_positions<<<n_block, BLOCK_SIZE>>>(raw_pointer_cast(device_m.data()),
                                                  raw_pointer_cast(device_pos.data()),
                                                  raw_pointer_cast(device_v.data()), n_body);

        // copy updated position back to host
        host_pos = device_pos;

        high_resolution_clock::time_point t2 = high_resolution_clock::now();
        duration<double> time_span = t2 - t1;
        std::cout << "Iteration " << i << ", elapsed time: " << time_span.count() << '\n';

#ifdef GUI
        glut_update(host_pos.data(), host_pos.size());
#endif
    }
}

int main(int argc, char* argv[]) {
    const std::string prog_name = "CUDA";

    n_body = std::stoi(argv[1]);
    n_iteration = std::stoi(argv[2]);

#ifdef GUI
    glut_init(argc, argv, prog_name);
#endif

    TIME_IT(master();)

    print_information(prog_name, n_body, total_threads);

    return 0;
}
