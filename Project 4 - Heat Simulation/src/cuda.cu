#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iomanip>
#include <iostream>

#include "common.h"

constexpr int BLOCK_SIZE = 512;

int num_cuda_threads;

__global__ void generate_fire_area_cuda(bool* device_fire_area, const int size, const int len) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= len) return;
    const int i = idx / size, j = idx % size;

    constexpr int fire1_r2 = FIRE_SIZE * FIRE_SIZE;
    const int a = i - size / 2;
    const int b = j - size / 2;
    const int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
    if (r2 < fire1_r2) device_fire_area[i * size + j] = true;

    constexpr int fire2_r2 = (FIRE_SIZE / 2) * (FIRE_SIZE / 2);
    const int c = i - 1 * size / 3;
    const int d = j - 1 * size / 3;
    const int r22 = c * c + d * d;
    if (r22 < fire2_r2) device_fire_area[i * size + j] = true;
}

__global__ void update_cuda(const float* device_data, float* device_new_data, 
                            const bool* device_fire_area, const int size, const int len) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx - size < 0 || idx + size >= len) return; // to avoid invalid array access

    if (device_fire_area[idx]) device_new_data[idx] = FIRE_TEMP;
    else {
        const float up = device_data[idx - size];
        const float down = device_data[idx + size];
        const float left = device_data[idx - 1];
        const float right = device_data[idx + 1];
        const float new_val = (up + down + left + right) / 4;
        device_new_data[idx] = new_val;
    }
}

__global__ void maintain_wall_cuda(float* device_new_data, const int size, const int len) {
    const int t_idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (t_idx >= size) return;

    device_new_data[t_idx] = WALL_TEMP;
    device_new_data[t_idx * size] = WALL_TEMP;
    device_new_data[t_idx * size + size - 1] = WALL_TEMP;
    device_new_data[len - size + t_idx] = WALL_TEMP;
}

void master() {
    float* device_data;
    float* device_new_data;
    bool* device_fire_area;
    bool* continue_arr;
    //! to copy a variable from device to host via hipMemcpy, you MUST allocate the variable via
    //! hipMalloc!
    bool* device_cont;

    initialize();

    // calculate number of CUDA threads to use
    const int num_blocks = len % BLOCK_SIZE ? len / BLOCK_SIZE + 1 : len / BLOCK_SIZE;
    num_cuda_threads = num_blocks * BLOCK_SIZE;

    hipMalloc(&device_data, len * sizeof(float));
    hipMalloc(&device_new_data, len * sizeof(float));
    hipMalloc(&device_fire_area, len * sizeof(bool));
    hipMalloc(&continue_arr, len * sizeof(bool));
    hipMalloc(&device_cont, sizeof(bool));

    generate_fire_area_cuda<<<num_blocks, BLOCK_SIZE>>>(device_fire_area, size, len);

    // copy data from host to device
    hipMemcpy(device_data, data.data(), len * sizeof(float), hipMemcpyHostToDevice);
    
    int count = 1;
    double total_time = 0;

    std::cout << std::fixed << std::setprecision(6);
    while (count <= MAX_ITERATIONS) {
        time_point t1 = std_clock::now();

        update_cuda<<<num_blocks, BLOCK_SIZE>>>(device_data, device_new_data, device_fire_area, size, len);
        maintain_wall_cuda<<<num_blocks, BLOCK_SIZE>>>(device_new_data, size, len);

        // copy data from device to host and swap data and new_data
#ifdef GUI
        hipMemcpy(data.data(), device_new_data, len * sizeof(float), hipMemcpyDeviceToHost);
#endif
        // flip data and new_data after one iteration to avoid copying data
        std::swap(device_data, device_new_data);

        time_point t2 = std_clock::now();
        const double time = std::chrono::duration<double>(t2 - t1).count();
        if (DEBUG) std::cout << "Iteration " << count << " ,elapsed time: " << time << '\n';
        total_time += time;

#ifdef GUI
        // plot(data);
        data2pixels(data, pixels);
        plot(pixels);
#endif

        count++;
    }

    std::cout << "Converge after " << count - 1 << " iterations, elapsed time: " << total_time
              << ", average computation time: " << total_time / (count - 1) << std::endl;

    hipFree(device_data);
    hipFree(device_new_data);
    hipFree(device_fire_area);
    hipFree(continue_arr);
    hipFree(&device_cont);
}

int main(const int argc, char* argv[]) {
    const std::string name = "CUDA";

    size = std::stoi(argv[1]);

#ifdef GUI
    glut_init(argc, argv, name);
#endif

    master();

    print_info(name, num_cuda_threads);

    return 0;
}
